#include "hip/hip_runtime.h"

#include "kernel.h"

    size_t width;
    size_t height;
    float* cudaInput;
    float* cudaOutputX;
    float* cudaOutputY;
    float* cudaOutputAux;
    float* gaussianKernelCuda;
    float* cudaSobelX;
    float* cudaSobelY;
    float* cudaA_X_X;
    float* cudaA_X_Y;
    float* cudaA_Y_Y;


__global__ void gaussianBlurKernel(const float* const __restrict__ input,
                                   float* const __restrict__ output,
                                   const size_t width,
                                   const size_t height,
                                   const float* const __restrict__ gaussianKernel)
{
//x and y maxs are width and height
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    Byte inputs[9];

    if((x > 0) && (x < (height - 1)) && (y > 0) && (y < (width - 1)))
    {
        inputs[0]  = input[(y - 1) * width + (x - 1)];
        inputs[1]  = input[(y - 1) * width + x];
        inputs[2]  = input[(y - 1) * width +  (x + 1)];
        inputs[3]  = input[y * width + (x - 1)];
        inputs[4]  = input[y * width + x];
        inputs[5]  = input[y * width + (x + 1)];
        inputs[6]  = input[(y + 1) * width + (x - 1)];
        inputs[7]  = input[(y + 1) * width + x];
        inputs[8]  = input[(y + 1) * width + (x + 1)];

        unsigned int tempValue = 0;
        for (unsigned int it = 0; it < 9; ++it)
              tempValue += inputs[it] * gaussianKernel[it];

        output[y * width + x] = (tempValue > 255)?255:tempValue;
    }
    else
        output[y * width + x] = 255;
};

__global__ void sobelKernel(const float* const __restrict__ input,
                            float* const __restrict__ outputX,
                            float* const __restrict__ outputY,
                            const size_t width,
                            const size_t height,
                            const float* const __restrict__ sobelKernelX,
                            const float* const __restrict__ sobelKernelY)
{
//x and y maxs are width and height
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float inputs[9];

    if((x > 0) && (x < (height - 1)) && (y > 0) && (y < (width - 1)))
    {
        inputs[0]  = input[(y - 1) * width + (x - 1)];
        inputs[1]  = input[(y - 1) * width + x];
        inputs[2]  = input[(y - 1) * width +  (x + 1)];
        inputs[3]  = input[y * width + (x - 1)];
        inputs[4]  = input[y * width + x];
        inputs[5]  = input[y * width + (x + 1)];
        inputs[6]  = input[(y + 1) * width + (x - 1)];
        inputs[7]  = input[(y + 1) * width + x];
        inputs[8]  = input[(y + 1) * width + (x + 1)];

        float tempValueX = 0.0f;
        float tempValueY = 0.0f;
        for (unsigned int it = 0; it < 9; ++it)
        {
              tempValueX += inputs[it] * sobelKernelX[it];
              tempValueY += inputs[it] * sobelKernelY[it];
        }

        outputX[y * width + x] = tempValueX;
        outputY[y * width + x] = tempValueY;
    }
    else
    {
        outputX[y * width + x] = 0.0f;
        outputY[y * width + x] = 0.0f;
    }
};

__global__ void cwiseProduct(const float* const matrix1,
                           const float* const matrix2,
                           float* const output,
                           const size_t width,
                           const size_t height)
{
//TODO Ejercicio 1 Hacer el kernel para hacer el producto punto a punto
}

void harrisCornersFilter(const float* const image,
                         const size_t imageWidth,
                         const size_t imageHeight,
                         const float* const gaussianKernel,
                         float* output)
{
//Inicializacion de memoria

    width = imageWidth;
    height = imageHeight;

    hipMalloc(reinterpret_cast<void**>(&cudaInput), width * height * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&gaussianKernelCuda), 9 * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&cudaSobelX), 9 * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&cudaSobelY), 9 * sizeof(float));

    hipMalloc(reinterpret_cast<void**>(&cudaOutputX), width * height * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&cudaOutputY), width * height * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&cudaOutputAux), width * height * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&cudaA_X_X), width * height * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&cudaA_X_Y), width * height * sizeof(float));
    hipMalloc(reinterpret_cast<void**>(&cudaA_Y_Y), width * height * sizeof(float));

    float sobelKernelX[] = {-1.0f, 0.0f, 1.0f,
                            -2.0f, 0.0f, 2.0f,
                            -1.0f, 0.0f, 1.0f
                           };

    float sobelKernelY[] = {-1.0f, -2.0f, -1.0f,
                             0.0f,  0.0f,  0.0f,
                             1.0f,  2.0f,  1.0f
                           };

    hipMemcpy(cudaInput, image, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gaussianKernelCuda, gaussianKernel, 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaSobelX, sobelKernelX, 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaSobelY, sobelKernelY, 9 * sizeof(float), hipMemcpyHostToDevice);

//Comienzo del calculo

    gradientSobelCuda();
    hipDeviceSynchronize();

    calculateA();

    //tip: Agregar outputs y usar streams

    gaussianBlurCuda(cudaA_X_X,cudaOutputAux);
    hipDeviceSynchronize();
    hipMemcpy(cudaA_X_X, cudaOutputAux, width * height * sizeof(float), hipMemcpyDeviceToDevice);

    gaussianBlurCuda(cudaA_X_Y,cudaOutputAux);
    hipDeviceSynchronize();
    hipMemcpy(cudaA_X_Y, cudaOutputAux, width * height * sizeof(float), hipMemcpyDeviceToDevice);

    gaussianBlurCuda(cudaA_Y_Y,cudaOutputAux);
    hipDeviceSynchronize();
    hipMemcpy(cudaA_Y_Y, cudaOutputAux, width * height * sizeof(float), hipMemcpyDeviceToDevice);

    calculateR();
    hipDeviceSynchronize();

    threshold();
    hipDeviceSynchronize();

    //Aqui dentro dejar en el rango [0, 1] a cada pixel de la imagen;
    nonMaximaSupression();
    hipDeviceSynchronize();

//copiamos el resultado
    hipMemcpy(output, cudaOutputAux, width * height * sizeof(float), hipMemcpyDeviceToHost);

//Liberamos memoria

    hipFree(cudaInput);
    hipFree(cudaOutputX);
    hipFree(cudaOutputY);
    hipFree(cudaOutputAux);
    hipFree(gaussianKernelCuda);
    hipFree(cudaSobelX);
    hipFree(cudaSobelY);
    hipFree(cudaA_X_X);
    hipFree(cudaA_X_Y);
    hipFree(cudaA_Y_Y);
}

void gaussianBlurCuda(const float* const input,
                      float* const output)
{
    dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 gridSize(width / BLOCK_SIZE_X, height / BLOCK_SIZE_Y);

    gaussianBlurKernel<<<gridSize, blockSize>>>(input,
                                                output,
                                                width,
                                                height,
                                                gaussianKernelCuda);
}

void gradientSobelCuda()
{
    dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 gridSize(width / BLOCK_SIZE_X, height / BLOCK_SIZE_Y);

    sobelKernel<<<gridSize, blockSize>>>(cudaInput,
                                         cudaOutputX,
                                         cudaOutputY,
                                         width,
                                         height,
                                         cudaSobelX,
                                         cudaSobelY);
}

void calculateA()
{
  dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y);
  dim3 gridSize(width / BLOCK_SIZE_X, height / BLOCK_SIZE_Y);

  //tip: Se pueden usar streams

  //cudaA_X_X = cudaOutputX * cudaOutputX;
  cwiseProduct<<<gridSize, blockSize>>>(cudaOutputX,
                                        cudaOutputX,
                                        cudaA_X_X,
                                        width,
                                        height);
  //cudaA_X_Y = cudaOutputX * cudaOutputY;
  cwiseProduct<<<gridSize, blockSize>>>(cudaOutputX,
                                        cudaOutputY,
                                        cudaA_X_Y,
                                        width,
                                        height);

  //cudaA_Y_Y = cudaOutputY * cudaOutputY;
  cwiseProduct<<<gridSize, blockSize>>>(cudaOutputY,
                                        cudaOutputY,
                                        cudaA_Y_Y,
                                        width,
                                        height);
}

void calculateR()
{
//TODO Ejercicio 2 hacer el kernel para calcular R con la siguiente funcion y llamarlo.
//tip: usar el kernel producto punto del ejercicio 1
    const float k = 0.04f;
//    R(cudaOutputAux) = cudaA_X_X * cudaA_Y_Y -
//                       cudaA_X_Y * cudaA_X_Y -
//                       k * (cudaA_X_X + cudaA_Y_Y) * (cudaA_X_X + cudaA_Y_Y);
}

void threshold()
{
//TODO Ejercicio 3 calcular el umbral a R (cudaOutputAux)
}

void nonMaximaSupression()
{
//TODO Ejercicio 4 calcular NMS a R (cudaOutputAux) y dejar R en el rango [0, 1]
}
